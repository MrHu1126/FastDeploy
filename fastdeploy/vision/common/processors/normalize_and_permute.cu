
#include <hip/hip_runtime.h>
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef WITH_GPU
#include "fastdeploy/vision/common/processors/normalize_and_permute.h"

namespace fastdeploy {
namespace vision {

__global__ void NormalizeAndPermuteKernel(const uint8_t* src, float* dst,
                                          const float* alpha, const float* beta,
                                          int num_channel, bool swap_rb,
                                          int batch_size, int edge) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= edge) return;

  int img_size = edge / batch_size;
  int n = idx / img_size;        // batch index
  int p = idx - (n * img_size);  // pixel index within the image

  // if (swap_rb) {
  //   uint8_t tmp = src[num_channel * idx];
  //   src[num_channel * idx] = src[num_channel * idx + 2];
  //   src[num_channel * idx + 2] = tmp;
  // }

  for (int i = 0; i < num_channel; ++i) {
    int j = i;
    if (swap_rb) {
      j = 2 - i;
    }
    dst[n * img_size * num_channel + i * img_size + p] =
        src[num_channel * idx + j] * alpha[i] + beta[i];
  }
}

bool NormalizeAndPermute::ImplByCuda(Mat* mat) {
  std::cout << "NormalizeAndPermute cuda" << std::endl;
  // Prepare input tensor
  FDTensor* src = CreateCachedGpuInputTensor(mat);

  // Prepare output tensor
  mat->output_cache->Resize(src->Shape(), FDDataType::FP32, "output_cache",
                            Device::GPU);

  // Copy alpha and beta to GPU
  gpu_alpha_.Resize({1, 1, static_cast<int>(alpha_.size())}, FDDataType::FP32,
                    "alpha", Device::GPU);
  cudaMemcpy(gpu_alpha_.Data(), alpha_.data(), gpu_alpha_.Nbytes(),
             cudaMemcpyHostToDevice);

  gpu_beta_.Resize({1, 1, static_cast<int>(beta_.size())}, FDDataType::FP32,
                   "beta", Device::GPU);
  cudaMemcpy(gpu_beta_.Data(), beta_.data(), gpu_beta_.Nbytes(),
             cudaMemcpyHostToDevice);

  int jobs = 1 * mat->Width() * mat->Height();
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);
  NormalizeAndPermuteKernel<<<blocks, threads, 0, mat->Stream()>>>(
      reinterpret_cast<uint8_t*>(src->Data()),
      reinterpret_cast<float*>(mat->output_cache->Data()),
      reinterpret_cast<float*>(gpu_alpha_.Data()),
      reinterpret_cast<float*>(gpu_beta_.Data()), mat->Channels(), swap_rb_, 1,
      jobs);

  mat->SetTensor(mat->output_cache);
  mat->device = Device::GPU;
  mat->layout = Layout::CHW;
  mat->mat_type = ProcLib::CUDA;
  return true;
}

bool NormalizeAndPermute::ImplByCuda(MatBatch* mat_batch) {
  std::cout << "NormalizeAndPermute cuda" << std::endl;

  // if (!CheckShapeConsistency(mats)) {
  //   return false;
  // }

  // Mat* mat = &(*(mat_batch->mats))[0];

  // Prepare input tensor
  // std::string tensor_name = Name() + "_cvcuda_src";
  FDTensor* src = CreateCachedGpuInputTensor(mat_batch);

  src->PrintInfo();

  // Prepare output tensor
  mat_batch->output_cache->Resize(src->Shape(), FDDataType::FP32,
                                  "output_cache", Device::GPU);
  // NHWC -> NCHW
  std::swap(mat_batch->output_cache->shape[1],
            mat_batch->output_cache->shape[3]);

  // Copy alpha and beta to GPU
  gpu_alpha_.Resize({1, 1, static_cast<int>(alpha_.size())}, FDDataType::FP32,
                    "alpha", Device::GPU);
  cudaMemcpy(gpu_alpha_.Data(), alpha_.data(), gpu_alpha_.Nbytes(),
             cudaMemcpyHostToDevice);

  gpu_beta_.Resize({1, 1, static_cast<int>(beta_.size())}, FDDataType::FP32,
                   "beta", Device::GPU);
  cudaMemcpy(gpu_beta_.Data(), beta_.data(), gpu_beta_.Nbytes(),
             cudaMemcpyHostToDevice);

  // cudaStreamSynchronize(mat->Stream());

  int jobs =
      mat_batch->output_cache->Numel() / mat_batch->output_cache->shape[1];
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);
  NormalizeAndPermuteKernel<<<blocks, threads, 0, mat_batch->Stream()>>>(
      reinterpret_cast<uint8_t*>(src->Data()),
      reinterpret_cast<float*>(mat_batch->output_cache->Data()),
      reinterpret_cast<float*>(gpu_alpha_.Data()),
      reinterpret_cast<float*>(gpu_beta_.Data()),
      mat_batch->output_cache->shape[1], swap_rb_,
      mat_batch->output_cache->shape[0], jobs);

  // cudaStreamSynchronize(mat->Stream());

  mat_batch->output_cache->PrintInfo();

  mat_batch->SetTensor(mat_batch->output_cache);
  mat_batch->device = Device::GPU;
  mat_batch->layout = MatBatchLayout::NCHW;
  mat_batch->mat_type = ProcLib::CUDA;
  mat_batch->has_batched_tensor = true;
  return true;
}

#ifdef ENABLE_CVCUDA
bool NormalizeAndPermute::ImplByCvCuda(Mat* mat) { return ImplByCuda(mat); }

bool NormalizeAndPermute::ImplByCvCuda(MatBatch* mat_batch) {
  return ImplByCuda(mat_batch);
}
#endif

}  // namespace vision
}  // namespace fastdeploy
#endif
