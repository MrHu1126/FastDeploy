#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fastdeploy/vision/common/processors/normalize_and_permute.h"

namespace fastdeploy {
namespace vision {

__global__ void NormalizeAndPermuteKernel(
    uint8_t* src, float* dst, const float* alpha, const float* beta,
    int num_channel, bool swap_rb, int edge) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= edge) return;

  if (swap_rb) {
    uint8_t tmp = src[num_channel * idx];
    src[num_channel * idx] = src[num_channel * idx + 2];
    src[num_channel * idx + 2] = tmp;
  }

  for (int i = 0; i < num_channel; ++i) {
    dst[idx + edge * i] = src[num_channel * idx + i] * alpha[i] + beta[i];
  }
}

bool NormalizeAndPermute::ImplByCuda(Mat* mat) {
  cv::Mat* im = mat->GetOpenCVMat();
  std::string buf_name = Name() + "_src";
  std::vector<int64_t> shape = {im->rows, im->cols, im->channels()};
  FDTensor* src = UpdateAndGetReusedBuffer(shape, im->type(), buf_name,
                                           Device::GPU);
  FDASSERT(hipMemcpy(src->Data(), im->ptr(), src->Nbytes(),
                      hipMemcpyHostToDevice) == 0,
           "Error occurs while copy memory from CPU to GPU.");

  buf_name = Name() + "_dst";
  FDTensor* dst = UpdateAndGetReusedBuffer(shape, CV_32FC(im->channels()),
                                           buf_name, Device::GPU);
  cv::Mat res(im->rows, im->cols, CV_32FC(im->channels()), dst->Data());

  buf_name = Name() + "_alpha";
  FDTensor* alpha = UpdateAndGetReusedBuffer({(int64_t)alpha_.size()}, CV_32FC1,
                                             buf_name, Device::GPU);
  FDASSERT(hipMemcpy(alpha->Data(), alpha_.data(), alpha->Nbytes(),
                      hipMemcpyHostToDevice) == 0,
           "Error occurs while copy memory from CPU to GPU.");

  buf_name = Name() + "_beta";
  FDTensor* beta = UpdateAndGetReusedBuffer({(int64_t)beta_.size()}, CV_32FC1,
                                             buf_name, Device::GPU);
  FDASSERT(hipMemcpy(beta->Data(), beta_.data(), beta->Nbytes(),
                      hipMemcpyHostToDevice) == 0,
           "Error occurs while copy memory from CPU to GPU.");

  int jobs = im->cols * im->rows;
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);
  NormalizeAndPermuteKernel<<<blocks, threads, 0, NULL>>>(
      reinterpret_cast<uint8_t*>(src->Data()),
      reinterpret_cast<float*>(dst->Data()),
      reinterpret_cast<float*>(alpha->Data()),
      reinterpret_cast<float*>(beta->Data()), im->channels(), swap_rb_, jobs);

  mat->SetMat(res);
  mat->device = Device::GPU;
  mat->layout = Layout::CHW;
  return true;
}

}  // namespace vision
}  // namespace fastdeploy
